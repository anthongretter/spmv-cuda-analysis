#include "hip/hip_runtime.h"
#include "gpu.cuh"
#include <math.h>

template<unsigned int unroll_factor>
__device__ __forceinline__ PRIM_T compute_unrolled_sum(volatile struct CSR *csr, VEC_T vec, int &j) {
    PRIM_T sum = 0;

    if constexpr (unroll_factor >= 16) sum += csr->val[j + 15] * vec[csr->col[j + 15]];
    if constexpr (unroll_factor >= 15) sum += csr->val[j + 14] * vec[csr->col[j + 14]];
    if constexpr (unroll_factor >= 14) sum += csr->val[j + 13] * vec[csr->col[j + 13]];
    if constexpr (unroll_factor >= 13) sum += csr->val[j + 12] * vec[csr->col[j + 12]];
    if constexpr (unroll_factor >= 12) sum += csr->val[j + 11] * vec[csr->col[j + 11]];
    if constexpr (unroll_factor >= 11) sum += csr->val[j + 10] * vec[csr->col[j + 10]];
    if constexpr (unroll_factor >= 10) sum += csr->val[j + 9] * vec[csr->col[j + 9]];
    if constexpr (unroll_factor >= 9)  sum += csr->val[j + 8] * vec[csr->col[j + 8]];
    if constexpr (unroll_factor >= 8)  sum += csr->val[j + 7] * vec[csr->col[j + 7]];
    if constexpr (unroll_factor >= 7)  sum += csr->val[j + 6] * vec[csr->col[j + 6]];
    if constexpr (unroll_factor >= 6)  sum += csr->val[j + 5] * vec[csr->col[j + 5]];
    if constexpr (unroll_factor >= 5)  sum += csr->val[j + 4] * vec[csr->col[j + 4]];
    if constexpr (unroll_factor >= 4)  sum += csr->val[j + 3] * vec[csr->col[j + 3]];
    if constexpr (unroll_factor >= 3)  sum += csr->val[j + 2] * vec[csr->col[j + 2]];
    if constexpr (unroll_factor >= 2)  sum += csr->val[j + 1] * vec[csr->col[j + 1]];
    if constexpr (unroll_factor >= 1)  sum += csr->val[j + 0] * vec[csr->col[j + 0]];

    return sum;
}


__global__ void SPMV_kernel(int row, int col, int n, void* ptr_matrix, VEC_T vec, VEC_T result) {
    struct CSR *csr = (struct CSR *)ptr_matrix;

    int thread_row = blockIdx.x * blockDim.x + threadIdx.x;

    if (thread_row < row) {
        PRIM_T sum = 0;
        int row_start = csr->row[thread_row];
        int row_end = csr->row[thread_row + 1];
        int j = row_start;

        for (; row_end - j > 16; j++) {
            sum += csr->val[j] * vec[csr->col[j]];
        }
        
        int remaining = row_end - j;
        switch (remaining) {
            case 16: sum += compute_unrolled_sum<16>(csr, vec, j); break;
            case 15: sum += compute_unrolled_sum<15>(csr, vec, j); break;
            case 14: sum += compute_unrolled_sum<14>(csr, vec, j); break;
            case 13: sum += compute_unrolled_sum<13>(csr, vec, j); break;
            case 12: sum += compute_unrolled_sum<12>(csr, vec, j); break;
            case 11: sum += compute_unrolled_sum<11>(csr, vec, j); break;
            case 10: sum += compute_unrolled_sum<10>(csr, vec, j); break;
            case 9:  sum += compute_unrolled_sum<9>(csr, vec, j); break;
            case 8:  sum += compute_unrolled_sum<8>(csr, vec, j); break;
            case 7:  sum += compute_unrolled_sum<7>(csr, vec, j); break;
            case 6:  sum += compute_unrolled_sum<6>(csr, vec, j); break;
            case 5:  sum += compute_unrolled_sum<5>(csr, vec, j); break;
            case 4:  sum += compute_unrolled_sum<4>(csr, vec, j); break;
            case 3:  sum += compute_unrolled_sum<3>(csr, vec, j); break;
            case 2:  sum += compute_unrolled_sum<2>(csr, vec, j); break;
            case 1:  sum += compute_unrolled_sum<1>(csr, vec, j); break;
            case 0:  break;
        }
        result[thread_row] = sum;
    }
}

void SPMV(int row, int col, int n, void* ptr_matrix, VEC_T vec, VEC_T result) {
    int BLOCKS_PER_GRID = (row + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    SPMV_kernel<<<BLOCKS_PER_GRID, THREADS_PER_BLOCK>>>(row, col, n, ptr_matrix, vec, result);
}

void* SPMV_setup(int row, int col, int n, MATRIX_T matrix) {
    struct CSR *host_csr = matrix_csr_format(row, col, n, matrix);

    struct CSR *device_csr;
    hipMalloc(&device_csr, sizeof(struct CSR));

    int *d_row, *d_col;
    PRIM_T *d_val;
    hipMalloc(&d_row, (row + 1) * sizeof(int));
    hipMalloc(&d_col, n * sizeof(int));
    hipMalloc(&d_val, n * sizeof(PRIM_T));

    hipMemcpyAsync(d_row, host_csr->row, (row + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_col, host_csr->col, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpyAsync(d_val, host_csr->val, n * sizeof(PRIM_T), hipMemcpyHostToDevice);

    struct CSR host_csr_copy = *host_csr;
    host_csr_copy.row = d_row;
    host_csr_copy.col = d_col;
    host_csr_copy.val = d_val;

    hipMemcpyAsync(device_csr, &host_csr_copy, sizeof(struct CSR), hipMemcpyHostToDevice);

    hipFree(host_csr->row);
    hipFree(host_csr->col);
    hipFree(host_csr->val);
    hipFree(host_csr);

    return (void*)device_csr;
}

void SPMV_free(void* ptr_matrix) {
    struct CSR host_csr;
    struct CSR *device_csr = (struct CSR *)ptr_matrix;

    hipMemcpyAsync(&host_csr, device_csr, sizeof(struct CSR), hipMemcpyDeviceToHost);

    hipFree(host_csr.row);
    hipFree(host_csr.col);
    hipFree(host_csr.val);
    hipFree(device_csr);
}


size_t SPMV_overall_accesses(int row, int col, int n) {
    return (row + 1) * sizeof(int) +  // csr->row
           n * sizeof(int) +          // csr->col
           col * sizeof(PRIM_T) +     // vec
           col * sizeof(PRIM_T) +     // result
           n * sizeof(PRIM_T);        // csr->val
}
